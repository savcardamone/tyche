#include "hip/hip_runtime.h"
/**
 * @file n_cody.cu
 * @author Salvatore Cardamone
 * @brief Routines for an N-Body simulation which can be ported to the GPU.
 */
#include "n_body/n_body.hpp"

/**
 * @brief Initialise e
 *
 * @param[in] nParticles Number of particles we're dealing with.
 */
__global__
void NBody::InitialiseDevice(const unsigned int& nParticles)
  : nParticles_(nParticles), particles(nParticles), acc(nParticles) {

}

/**
 * @brief
 *
 */
__global__
void NBody::CalculateForces() {

  extern __shared__ float4[] shared_positions;

  // Get the index of the particle the thread is going to be working on
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  auto thread_particle = particles[tid];

  //
  for (auto i = 0, iTile = 0; i < nParticles; i += nThreadWarp, ++iTile) {

    // Every thread loads a shared position into the shared array
    int idx = tile * blockDim.x + threadIdx.x;
    shared_positions[threadIdx.x] = particles[idx];
    __syncthreads();

    //
    acc = tile_calculation(myPosition, acc);
    __syncthreads();

  }

  // Save the result in global memory for the integration step.
  float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
  globalA[gtid] = acc4;

}

/**
 * @brief Update the acceleration of particle i as a result of its interaction
 *        with particle j. 19 FLOPs if we count the inverse square root as a
 *        single FLOP, 20 otherwise.
 * @param i Position and mass of particle i.
 * @param j Position and mass of particle j.
 * @param acc_i Acceleration of particle i, updated on return.
 */
__device__
void NBody::PairwiseInteraction(const float4& i, const float4& j,
                                float3& acc_i) {

  // Vector difference between the two particles
  float3 rij;
  rij.x = j.x - i.x;
  rij.y = j.y - i.y;
  rij.z = j.z - i.z;

  // Compute the distance-dependent component of the force
  // Note that we don't need the product of masses since we're skipping the
  // force calculation and going straight to acceleration
  float r2 = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z + eps2;
  float r6 = r2 * r2 * r2;
  float temp = j.w  / sqrt(r6);

  // Update acceleration: Do FMA so we get two FLOPs for the price of one
  acc_i.x += rij.x * temp;
  acc_i.y += rij.y * temp;
  acc_i.z += rij.z * temp;

}

/**
 * @brief Compute the accelerations of the particles in a tile with all others
 *        in the system.
 * @param i Position and mass of particle i.
 */
__device__
void NBody::CalculateTile(const float4& i) {

  // The tile has a shared set of particles. Each thread involved in computing
  // the tile iterates over this shared set of particles
  extern __shared__ float4[] shared_particles;

  // For every particle in the shared particles, compute interaction with
  // particle assigned to thread
  for (auto j=0; j<blockDim.x; ++j) {
    PariwiseInteraction(i, shared_particles[j], acc[i]);
  }

}
